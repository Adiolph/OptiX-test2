#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

struct PerRayData_pathtrace
{
  unsigned int hitID;
  unsigned int seed;
};

rtDeclareVariable(float3, hit_pos, attribute hit_pos, );
rtDeclareVariable(PerRayData_pathtrace, prd, rtPayload, );

RT_PROGRAM void closest_hit()
{
  prd.hitID = hit_pos.x / 10;
};


