#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

struct PerRayData_pathtrace
{
  unsigned int hitID;
  unsigned int seed;
};

rtDeclareVariable(float3, hit_pos, attribute hit_pos, );
rtDeclareVariable(PerRayData_pathtrace, prd_radiance, rtPayload, );

RT_PROGRAM void closest_hit_radiance()
{
  prd_radiance.hitID = hit_pos.x / 10;
};


