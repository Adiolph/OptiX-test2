#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "PerRayData_pathtrace.h"

using namespace optix;

rtDeclareVariable(float3, hit_pos, attribute hit_pos, );
rtDeclareVariable(PerRayData_pathtrace, prd, rtPayload, );

RT_PROGRAM void closest_hit()
{
  prd.hitID = hit_pos.x / 10;
};
