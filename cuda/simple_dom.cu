#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "PerRayData_pathtrace.h"

using namespace optix;

rtDeclareVariable(float3, hit_pos, attribute hit_pos, );
rtDeclareVariable(PerRayData_pathtrace, prd, rtPayload, );

RT_PROGRAM void closest_hit()
{
  prd.hitID = __float2int_rd(hit_pos.x);
  rtPrintf("hitID: %d, hit_pos: %.3f, %.3f, %.3f \n", prd.hitID, hit_pos.x, hit_pos.y, hit_pos.z);
};
