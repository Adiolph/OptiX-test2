#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "random.h"  // OptiX random header file in SDK/cuda/random.h
#include "PerRayData_pathtrace.h"
#include "cherenkov_step.h"
#include "rotateUz.h"

using namespace optix;

#define PI 3.1415926

rtDeclareVariable(unsigned int, random_seed, , );  // the random seed of the kernel
rtDeclareVariable(rtObject, top_object, , );  // group object
rtDeclareVariable(uint, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, launch_dim, rtLaunchDim, );
rtBuffer<CherenkovStep> cherenkov_steps;  // the step that can emit Cherenkov photons
rtBuffer<uint, 1> output_id;  // record the id of dom that photon hit, 0 if no hit

__device__ float3 gen_CK_dir(float costh, float sinth, unsigned int &random_seed);

RT_PROGRAM void gen_cherenkov()
{
  const CherenkovStep cherenkov_step = cherenkov_steps[0];
  PerRayData_pathtrace prd;
  prd.seed = tea<4>(launch_index, random_seed);
  prd.command = 0;
  prd.hitID = 0;
  float3 ray_origin = cherenkov_step.pos + cherenkov_step.length * rnd(prd.seed) * cherenkov_step.dir;
  // TODO: add wave length depedented cherenkov angle
  float3 ray_dir = gen_CK_dir(sqrtf(3)/2, 0.5, prd.seed);
  rotateUz(ray_dir, cherenkov_step.dir);
  rtPrintf("// point_source  ray_direction: (%.3f %.3f %.3f), ray_origin: (%.3f %.3f %.3f) \n",
    ray_dir.x, ray_dir.y, ray_dir.z,
    ray_origin.x, ray_origin.y, ray_origin.z
  );

  Ray ray = make_Ray(ray_origin, ray_dir, 0, 0.01, RT_DEFAULT_MAX);
  rtTrace(top_object, ray, prd);

  int flag_continue = 1;
  int count = 0;
  while(flag_continue)
  {
    // test scattering, absorption and hit.
    switch(prd.command)
    {
      // scattering: generate a new direction according to original direction, re-emit rays
      case SCA:
      {
        float3 SCA_dir = gen_CK_dir(sqrtf(1-0.01), 0.1, prd.seed);
        ray_origin = ray_origin + prd.length * ray_dir;
        rotateUz(ray_dir, SCA_dir);
        ray = make_Ray(ray_origin, ray_dir, 0, 0.01, RT_DEFAULT_MAX);
        prd.length = 0;
        prd.command = 0;
        rtTrace(top_object, ray, prd);
        break;
      }
      case ABS:
      {
        count++;
        ray_origin = cherenkov_step.pos + cherenkov_step.length * rnd(prd.seed) * cherenkov_step.dir;
        ray_dir = gen_CK_dir(sqrtf(3)/2, 0.5, prd.seed);
        rotateUz(ray_dir, cherenkov_step.dir);
        ray = make_Ray(ray_origin, ray_dir, 0, 0.01, RT_DEFAULT_MAX);
        prd.length = 0;
        prd.command = 0;
        rtTrace(top_object, ray, prd);
        break;
      }
      case HIT:
      {
        output_id[launch_index] = prd.hitID;
        flag_continue = 0;
        break;
      }
    }
  }
  rtPrintf("hitID: %d, num_photon: %d. \n", prd.hitID, count);
}

__device__ float3 gen_CK_dir(float cosTheta, float sinTheta, unsigned int &random_seed)
{
    float3 dir;
    float phi = rnd(random_seed) * 2 * M_PI;
    float sinPhi, cosPhi;
    sincosf(phi, &sinPhi, &cosPhi);
    dir.x = cosPhi * sinTheta;
    dir.y = sinPhi * sinTheta;
    dir.z = cosTheta;
    return dir;
}

rtDeclareVariable(PerRayData_pathtrace, prd, rtPayload, );

RT_PROGRAM void exception()
{
  rtPrintExceptionDetails();
}

RT_PROGRAM void miss()
{
  float Len_Abs = 30;
  float Len_Sca = 50;
  float len_abs = -Len_Abs * logf(rnd(prd.seed));
  float len_sca = -Len_Sca * logf(rnd(prd.seed));
  if (len_sca < len_abs)
  {
    prd.command = SCA;
    prd.length = len_sca;
  }
  else
  {
    prd.command = ABS;
  }

}
