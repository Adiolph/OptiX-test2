#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "random.h"  // OptiX random header file in SDK/cuda/random.h
#include "PerRayData_pathtrace.h"
#include "cherenkov_step.h"

using namespace optix;

#define PI 3.1415926

rtDeclareVariable(unsigned int, random_seed, , );  // the random seed of the kernel
rtDeclareVariable(rtObject, top_object, , );  // group object
rtDeclareVariable(uint, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, launch_dim, rtLaunchDim, );
rtBuffer<CherenkovStep> cherenkov_steps;  // the step that can emit Cherenkov photons
rtBuffer<uint, 1> output_id;  // record the id of dom that photon hit, 0 if no hit

__device__ float3 gen_CK_dir(float costh, float sinth, unsigned int &random_seed);
__device__ float3 rotate_by_axis(float3 axis, float3 vec);

RT_PROGRAM void gen_cherenkov()
{
  CherenkovStep cherenkov_step = cherenkov_steps[0];
  PerRayData_pathtrace prd;
  prd.seed = tea<4>(launch_index, random_seed);
  prd.command = 0;
  prd.hitID = 0;
  float3 ray_origin = cherenkov_step.pos + cherenkov_step.length * rnd(prd.seed) * cherenkov_step.dir;
  float3 CK_dir = gen_CK_dir(sqrtf(3)/2, 0.5, prd.seed);
  float3 ray_dir = rotate_by_axis(cherenkov_step.dir, CK_dir);
  rtPrintf("//point_source  ray_direction (%.3f %.3f %.3f) ray_origin (%.3f %.3f %.3f)  \n", 
  ray_dir.x, ray_dir.y, ray_dir.z,
  ray_origin.x, ray_origin.y, ray_origin.z
  );  

  Ray ray = make_Ray(ray_origin, ray_dir, 0, 0.01, RT_DEFAULT_MAX);
  rtTrace(top_object, ray, prd);

  int flag_continue = 1;
  int count = 0;
  while(flag_continue)
  {
    // test scattering, absorption and hit.
    switch(prd.command)
    {
      // scattering: generate a new direction according to original direction, re-emit rays
      case SCA:
      {
        float3 SCA_dir = gen_CK_dir(sqrtf(1-0.01), 0.1, prd.seed);
        ray_origin = ray_origin + prd.length * ray_dir;
        ray_dir = rotate_by_axis(ray_dir, SCA_dir);
        ray = make_Ray(ray_origin, ray_dir, 0, 0.01, RT_DEFAULT_MAX);
        prd.length = 0;
        prd.command = 0;
        rtTrace(top_object, ray, prd);
        break;
      }
      case ABS:
      {
        count++;
        CK_dir = gen_CK_dir(sqrtf(3)/2, 0.5, prd.seed);
        ray_dir = rotate_by_axis(cherenkov_step.dir, CK_dir);
        ray_origin = cherenkov_step.pos + cherenkov_step.length * rnd(prd.seed) * cherenkov_step.dir;
        ray = make_Ray(ray_origin, ray_dir, 0, 0.01, RT_DEFAULT_MAX);
        prd.length = 0;
        prd.command = 0;
        rtTrace(top_object, ray, prd);
        break;
      }
      case HIT:
      {
        output_id[launch_index] = prd.hitID;
        flag_continue = 0;
        break;
      }
    }
  }
  rtPrintf("hitID: %d, num_photon: %d. \n", prd.hitID, count);
}

__device__ float3 gen_CK_dir(float costh, float sinth, unsigned int &random_seed)
{
    float3 dir;
    double phi = rnd(random_seed) * 2 * M_PI;
    dir.x = cos(phi) * sinth;
    dir.y = sin(phi) * sinth;
    dir.z = costh;
    return dir;
}

__device__ float3 rotate_by_axis(float3 axis, float3 vec)
{
  float2 proj = make_float2(-axis.y, axis.x);
  proj /= sqrtf(axis.x*axis.x + axis.y*axis.y);
  float3 vec_new;
  // rotate vec by proj with angle of arccos(axis.z)
  vec_new.x = (axis.z+(1-axis.z)*proj.x*proj.x) * vec.x 
              + ((1-axis.z)*proj.x*proj.y) * vec.y 
              + (sqrtf(1-axis.z*axis.z)*proj.y) * vec.z;
  vec_new.y = ((1-axis.z)*proj.x*proj.y) * vec.x 
              + (axis.z+(1-axis.z)*proj.y*proj.y) * vec.y 
              + (-sqrtf(1-axis.z*axis.z)*proj.x) * vec.z;
  vec_new.z = (-sqrtf(1-axis.z*axis.z)*proj.y) * vec.x 
              + (sqrtf(1-axis.z*axis.z)*proj.x) * vec.y 
              + (axis.z) * vec.z;
  return vec_new;
}

rtDeclareVariable(PerRayData_pathtrace, prd, rtPayload, );

RT_PROGRAM void exception()
{
  rtPrintExceptionDetails();
}

RT_PROGRAM void miss()
{
  float Len_Abs = 30;
  float Len_Sca = 50;
  float len_abs = -Len_Abs * logf(rnd(prd.seed));
  float len_sca = -Len_Sca * logf(rnd(prd.seed));
  if (len_sca < len_abs)
  {
    prd.command = SCA;
    prd.length = len_sca;
  }
  else
  {
    prd.command = ABS;
  }

}
