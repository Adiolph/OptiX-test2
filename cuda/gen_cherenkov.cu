#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "random.h"  // OptiX random header file in SDK/cuda/random.h
#include "PerRayData_pathtrace.h"
#include "cherenkov_step.h"

using namespace optix;

#define PI 3.1415926

rtDeclareVariable(unsigned int, random_seed, , );  // the random seed of the kernel
rtDeclareVariable(CherenkovStep, cherenkov_step, , );  // the step that can emit Cherenkov photons
rtDeclareVariable(rtObject, top_object, , );  // group object
rtDeclareVariable(uint, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, launch_dim, rtLaunchDim, );
rtBuffer<uint, 1> output_id;  // record the id of dom that photon hit, 0 if no hit
rtBuffer<uint, 1> output_counts;  // record the counts of photon re-emission

__device__ float3 gen_CK_dir(float costh, float sinth, unsigned int &random_seed);
__device__ float3 rotate_by_axis(float3 axis, float3 vec);

RT_PROGRAM void gen_cherenkov()
{
  PerRayData_pathtrace prd;
  prd.seed = tea<4>(launch_index, random_seed);
  prd.command = 0;
  prd.hitID = 0;
  float3 ray_origin = cherenkov_step.pos + cherenkov_step.length * rnd(prd.seed) * cherenkov_step.dir;
  float3 CK_dir = gen_CK_dir(sqrtf(3)/2, 0.5, prd.seed);
  float3 ray_dir = rotate_by_axis(cherenkov_step.dir, CK_dir);
  Ray ray = make_Ray(ray_origin, ray_dir, 0, 0.01, RT_DEFAULT_MAX);
  rtTrace(top_object, ray, prd);


  int flag_nohit = 1;
  int count = 0;
  while(flag_nohit)
  {
    // test scattering, absorption and hit.
    switch(prd.command)
    {
      // scattering: generate a new direction according to original direction, re-emit rays
      case SCA:
      {
        float3 SCA_dir = gen_CK_dir(sqrtf(1-0.01), 0.1, prd.seed);
        ray_origin = ray_origin + prd.length * ray_dir;
        ray_dir = rotate_by_axis(ray_dir, SCA_dir);
        ray = make_Ray(ray_origin, ray_dir, 0, 0.01, RT_DEFAULT_MAX);
        prd.length = 0;
        prd.command = 0;
        rtTrace(top_object, ray, prd);
        break;
      }
      case ABS:
      {
        count++;
        CK_dir = gen_CK_dir(sqrtf(3)/2, 0.5, prd.seed);
        ray_dir = rotate_by_axis(cherenkov_step.dir, CK_dir);
        ray_origin = cherenkov_step.pos + cherenkov_step.length * rnd(prd.seed) * cherenkov_step.dir;
        ray = make_Ray(ray_origin, ray_dir, 0, 0.01, RT_DEFAULT_MAX);
        prd.length = 0;
        prd.command = 0;
        rtTrace(top_object, ray, prd);
        break;
      }
      case HIT:
      {
        output_id[launch_index] = prd.hitID;
        flag_nohit = 0;
        break;
      }
    }
  }
}

__device__ float3 gen_CK_dir(float costh, float sinth, unsigned int &random_seed)
{
    float3 dir;
    double phi = rnd(random_seed) * 2 * M_PI;
    dir.x = cos(phi) * sinth;
    dir.y = sin(phi) * sinth;
    dir.z = costh;
    return dir;
}

__device__ float3 rotate_by_axis(float3 axis, float3 vec)
{
  float2 proj = make_float2(-axis.y, axis.x);
  proj /= sqrtf(axis.x*axis.x + axis.y*axis.y);
  float3 vec_new;
  // rotate vec by proj with angle of arccos(axis.z)
  vec_new.x = (axis.z+(1-axis.z)*proj.x*proj.x) * vec.x 
              + ((1-axis.z)*proj.x*proj.y) * vec.y 
              + (sqrtf(1-axis.z*axis.z)*proj.y) * vec.z;
  vec_new.y = ((1-axis.z)*proj.x*proj.y) * vec.x 
              + (axis.z+(1-axis.z)*proj.y*proj.y) * vec.y 
              + (-sqrtf(1-axis.z*axis.z)*proj.x) * vec.z;
  vec_new.z = (-sqrtf(1-axis.z*axis.z)*proj.y) * vec.x 
              + (sqrtf(1-axis.z*axis.z)*proj.x) * vec.y 
              + (axis.z) * vec.z;
  return vec_new;
}

rtDeclareVariable(PerRayData_pathtrace, prd, rtPayload, );

RT_PROGRAM void exception()
{
    rtPrintExceptionDetails();
}

RT_PROGRAM void miss()
{
    prd.hitID = 42;
}
