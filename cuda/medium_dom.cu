#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include "PerRayData_pathtrace.h"

using namespace optix;

rtDeclareVariable(float3, hit_pos, attribute hit_pos, );
rtDeclareVariable(PerRayData_pathtrace, prd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, len_intersect, rtIntersectionDistance, );

RT_PROGRAM void closest_hit()
{
  float Len_Abs = 30;
  float Len_Sca = 50;
  float len_abs = -Len_Abs * logf(prd.seed);
  float len_sca = -Len_Sca * logf(prd.seed);
  if (len_abs < len_intersect)
  {
    if (len_sca < len_abs)
    {
      prd.command = SCA;
      prd.length = len_sca;
    }
    else
    {
      prd.command = ABS;
    }
  }
  else if(len_sca < len_intersect)
  {
    prd.command = SCA;
    prd.length = len_sca;
  }
  else
  {
    prd.command = HIT;
    prd.hitID = hit_pos.x / 10;
    rtPrintf("hitID: %d, hit_pos: %.3f, %.3f, %.3f \n", prd.hitID, hit_pos.x, hit_pos.y, hit_pos.z);
  }
};
