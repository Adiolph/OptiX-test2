#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include "PerRayData_pathtrace.h"
#include "random.h"  // OptiX random header file in SDK/cuda/random.h

using namespace optix;

rtDeclareVariable(float3, hit_pos, attribute hit_pos, );
rtDeclareVariable(PerRayData_pathtrace, prd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, len_intersect, rtIntersectionDistance, );

RT_PROGRAM void closest_hit()
{
  float Len_Abs = 30;
  float Len_Sca = 50;
  float len_abs = -Len_Abs * logf(rnd(prd.seed));
  float len_sca = -Len_Sca * logf(rnd(prd.seed));
  if (len_abs < len_intersect)
  {
    if (len_sca < len_abs)
    {
      prd.command = SCA;
      prd.length = len_sca;
    }
    else
    {
      prd.command = ABS;
    }
  }
  else if(len_sca < len_intersect)
  {
    prd.command = SCA;
    prd.length = len_sca;
  }
  else
  {
    prd.command = HIT;
    prd.hitID = __float2int_rd(hit_pos.x);
    rtPrintf("hitID: %d, hit_pos: %.3f, %.3f, %.3f \n", prd.hitID, hit_pos.x, hit_pos.y, hit_pos.z);
  }
};
