#include "hip/hip_runtime.h"
#include <optix_world.h>
using namespace optix;

struct PerRayData_camera
{
  float3 result;
  float  importance;
  int    depth;
};

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float3,        bg_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtBuffer<uchar4, 2>              output_buffer;
rtDeclareVariable(rtObject,      top_object, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 

// copied from OptiX SDK helpers.h
// Convert a float3 in [0,1)^3 to a uchar4 in [0,255]^4 -- 4th channel is set to 255
#ifdef __HIPCC__
static __device__ __inline__ optix::uchar4 make_color(const optix::float3& c)
{
    return optix::make_uchar4( static_cast<unsigned char>(__saturatef(c.z)*255.99f),  /* B */
                               static_cast<unsigned char>(__saturatef(c.y)*255.99f),  /* G */
                               static_cast<unsigned char>(__saturatef(c.x)*255.99f),  /* R */
                               255u);                                                 /* A */
}
#endif

RT_PROGRAM void ray_gen_camera()
{
  float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);
  
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, 0u, scene_epsilon, RT_DEFAULT_MAX);

  PerRayData_camera prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd ) ;
  output_buffer[launch_index] = make_color( prd.result );
}

RT_PROGRAM void exception_camera()
{
  rtPrintExceptionDetails();
  output_buffer[launch_index] = make_color( bad_color );
}

rtDeclareVariable(PerRayData_camera, prd, rtPayload, );

RT_PROGRAM void miss_camera()
{
  prd.result = bg_color;
}

RT_PROGRAM void closest_hit_camera()
{
  prd.result = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal))*0.5f + 0.5f;
}
